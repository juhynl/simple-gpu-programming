#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include "measure_host_time.h"

#define NUM_ITER_FOR_TEST 100

#define RAND_UNIFORM ((float)rand() / RAND_MAX)
#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))

double HW1_SPHERE_host(int n);
double HW1_SPHERE_reduce1(int n);
double HW1_SPHERE_thrust(int n);

__global__ void init_curand_states(hiprandState *states, unsigned long seed);
__global__ void init_curand_states(hiprandState *states, unsigned long *seeds, int n);
__global__ void do_MonteCarlo_simulation(hiprandState *states, float *counts, int n);
__global__ void reduce1(float *x, int n);

int main()
{
    int N = 1 << 24;                                                      // Number of points
    double unit_sphere_volume_exact = 4.0 / 3.0 * 3.14159265358979323846; // Exact volume of the sphere
    double unit_sphere_volume_simulated;
    float total_time_ms;

    // HW1_SPHERE_host
    // CHECK_TIME_START(_start, _freq);
    total_time_ms = 0;
    for (int i = 0; i < NUM_ITER_FOR_TEST; i++)
    {
        CHECK_TIME_START(_start);
        unit_sphere_volume_simulated = HW1_SPHERE_host(N);
        CHECK_TIME_END(_start, _end, _compute_time);
        total_time_ms += _compute_time;
    }
    _compute_time = total_time_ms / NUM_ITER_FOR_TEST;
    // CHECK_TIME_START(_start);
    // unit_sphere_volume_simulated = HW1_SPHERE_host(N);
    // // CHECK_TIME_END(_start, _end, _freq, _compute_time);
    // CHECK_TIME_END(_start, _end, _compute_time);
    fprintf(stdout, "\nHW1_SPHERE_host\nArea of unit sphere: ");
    fprintf(stdout, "simulated = %.15f / ", unit_sphere_volume_simulated);
    fprintf(stdout, "exact = %.15f / ", unit_sphere_volume_exact);
    fprintf(stdout, "relative error = %.15f\n", fabs(unit_sphere_volume_simulated - unit_sphere_volume_exact) / unit_sphere_volume_exact);
    fprintf(stdout, "*** Time to estimate the volume of a sphere = %.3f(ms)\n", _compute_time);

    // HW1_SPHERE_reduce1
    total_time_ms = 0;
    for (int i = 0; i < NUM_ITER_FOR_TEST; i++)
    {
        CHECK_TIME_START(_start);
        unit_sphere_volume_simulated = HW1_SPHERE_reduce1(N);
        CHECK_TIME_END(_start, _end, _compute_time);
        total_time_ms += _compute_time;
    }
    _compute_time = total_time_ms / NUM_ITER_FOR_TEST;
    // CHECK_TIME_START(_start);
    // unit_sphere_volume_simulated = HW1_SPHERE_reduce1(N);
    // CHECK_TIME_END(_start, _end, _compute_time);
    fprintf(stdout, "\nHW1_SPHERE_reduce1\nArea of unit sphere: ");
    fprintf(stdout, "simulated = %.15f / ", unit_sphere_volume_simulated);
    fprintf(stdout, "exact = %.15f / ", unit_sphere_volume_exact);
    fprintf(stdout, "relative error = %.15f\n", fabs(unit_sphere_volume_simulated - unit_sphere_volume_exact) / unit_sphere_volume_exact);
    fprintf(stdout, "*** Time to estimate the volume of a sphere = %.3f(ms)\n", _compute_time);

    // HW1_SPHERE_thrust
    total_time_ms = 0;
    for (int i = 0; i < NUM_ITER_FOR_TEST; i++)
    {
        CHECK_TIME_START(_start);
        unit_sphere_volume_simulated = HW1_SPHERE_thrust(N);
        CHECK_TIME_END(_start, _end, _compute_time);
        total_time_ms += _compute_time;
    }
    _compute_time = total_time_ms / NUM_ITER_FOR_TEST;
    // CHECK_TIME_START(_start);
    // unit_sphere_volume_simulated = HW1_SPHERE_thrust(N);
    // CHECK_TIME_END(_start, _end, _compute_time);
    fprintf(stdout, "\nHW1_SPHERE_reduce1\nArea of unit sphere: ");
    fprintf(stdout, "simulated = %.15f / ", unit_sphere_volume_simulated);
    fprintf(stdout, "exact = %.15f / ", unit_sphere_volume_exact);
    fprintf(stdout, "relative error = %.15f\n", fabs(unit_sphere_volume_simulated - unit_sphere_volume_exact) / unit_sphere_volume_exact);
    fprintf(stdout, "*** Time to estimate the volume of a sphere = %.3f(ms)\n", _compute_time);
}

double HW1_SPHERE_host(int n)
{
    srand(0);

    // Count of points inside the sphere
    int sum = 0;

    // Generate points and count those inside the sphere
    for (int i = 0; i < n; i++)
    {
        // Generate random points in the range [-1.0, 1.0]
        float x = 2.0f * RAND_UNIFORM - 1.0f;
        float y = 2.0f * RAND_UNIFORM - 1.0f;
        float z = 2.0f * RAND_UNIFORM - 1.0f;

        if (x * x + y * y + z * z <= 1.0f)
            sum++;
    }

    // Return the volume of the sphere
    return 8.0 * sum / n; // The volume of the unit cube is 8
}

double HW1_SPHERE_reduce1(int n)
{
    int threads = 256;
    int blocks = CEIL_DIV(n, threads); // assume N is a multiple of thread block size.

    // Initialize CURAND states
    hiprandState *d_states;
    hipMalloc(&d_states, n * sizeof(hiprandState));

    init_curand_states<<<blocks, threads>>>(d_states, time(NULL));
    hipDeviceSynchronize();

    // Execute Monte Carlo simulation kernel
    float *d_counts;
    hipMalloc(&d_counts, n * sizeof(double));

    do_MonteCarlo_simulation<<<blocks, threads, threads * sizeof(float)>>>(d_states, d_counts, n);
    hipDeviceSynchronize();

    // Reduce the simulation results
    reduce1<<<blocks, threads>>>(d_counts, n);
    reduce1<<<1, threads>>>(d_counts, blocks * threads);
    reduce1<<<1, 1>>>(d_counts, threads);
    hipDeviceSynchronize();

    // Copy final result back to host
    float sum;
    hipMemcpy(&sum, d_counts, sizeof(float), hipMemcpyDeviceToHost);

    // Free allocated device memory
    hipFree(d_states);
    hipFree(d_counts);

    // Return the volume of the sphere if (tid >= N)
    return 8.0 * sum / n; // The volume of the unit cube is 8
}

double HW1_SPHERE_thrust(int n)
{
    int threads = 256;
    int blocks = CEIL_DIV(n, threads); // assume N is a multiple of thread block size.

    // Initialize CURAND states
    hiprandState *d_states;
    hipMalloc(&d_states, n * sizeof(hiprandState));

    init_curand_states<<<blocks, threads>>>(d_states, time(NULL));
    hipDeviceSynchronize();

    // Execute Monte Carlo simulation kernel
    float *d_counts;
    hipMalloc(&d_counts, n * sizeof(double));

    do_MonteCarlo_simulation<<<blocks, threads, threads * sizeof(float)>>>(d_states, d_counts, n);
    hipDeviceSynchronize();

    // Reduce the simulation results
    thrust::device_vector<float> d_vec_counts(d_counts, d_counts + n);
    float sum = thrust::reduce(d_vec_counts.begin(), d_vec_counts.end());

    // Free allocated device memory
    hipFree(d_states);
    hipFree(d_counts);

    // Return the volume of the sphere
    return 8.0 * sum / n; // The volume of the unit cube is 8
}

__global__ void init_curand_states(hiprandState *states, unsigned long seed)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // each thread gets a different state
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void do_MonteCarlo_simulation(hiprandState *states, float *counts, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if the thread ID is within bounds
    if (tid >= n)
        return;

    // Initialize the count to 0
    counts[tid] = 0;

    // Get the state for this thread
    hiprandState localState = states[tid];

    // Generate a random float between -1.0 and 1.0
    float x = 2.0f * hiprand_uniform(&localState) - 1.0f;
    float y = 2.0f * hiprand_uniform(&localState) - 1.0f;
    float z = 2.0f * hiprand_uniform(&localState) - 1.0f;

    // Check if the generated point is inside the unit sphere
    if (x * x + y * y + z * z <= 1.0f)
        counts[tid] = 1;

    // Save the state back: unnecessary unless this kernel is called again
    states[tid] = localState;
}

__global__ void reduce1(float *x, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if the thread ID is within bounds
    if (tid >= n)
        return;

    // Perform a summation
    float tsum = 0.0f;
    int stride = gridDim.x * blockDim.x;
    for (int k = tid; k < n; k += stride)
    {
        tsum += x[k];
    }

    // Store the partial sum in the corresponding position of x
    x[tid] = tsum;
}