#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hipblas.h>

#include "measure_host_time.h"

#define FILE_A "my_file_A.bin"
#define FILE_B "my_file_B.bin"
#define FILE_A_HF "my_file_A_hf.bin"
#define FILE_B_HF "my_file_B_hf.bin"
#define FILE_C_1 "my_file_C_1.bin"
#define FILE_C_2 "my_file_C_2.bin"
#define FILE_C_3 "my_file_C_3.bin"
#define FILE_C_4 "my_file_C_4.bin"
#define FILE_C_5 "my_file_C_5.bin"
#define FILE_C_6 "my_file_C_6.bin"
#define FILE_C_7 "my_file_C_7.bin"

#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))

#define NUM_REPEATS 100

using namespace nvcuda;

// File read/write functions
void fread_matrix(const char *filename, float **matrix, int *row, int *col);
void fread_matrix(const char *filename, __half **matrix, int *row, int *col);
void fwrite_matrix(const char *filename, float *matrix, int row, int col);

// Matrix multiplication kernels
__global__ void mm_naive_cc(const float *A, const float *B, float *C, const int M, const int N, const int K);

template <const uint TS>
__global__ void mm_sm_cc(const float *A, const float *B, float *C, const int M, const int N, const int K);

template <const uint TS, const uint WPT, const uint RTS>
__global__ void mm_sm_mwpt_cc(const float *A, const float *B, float *C, const int M, const int N, const int K);

template <const uint WMMA_M, const uint WMMA_N, const uint WMMA_K>
__global__ void mm_naive_tc(__half *A, __half *B, float *C, const int M, const int N, const int K);

template <const uint TILE_M, const uint TILE_N, const uint TILE_K, const uint WMMA_M, const uint WMMA_N, const uint WMMA_K>
__global__ void mm_sm_tc(__half *A, __half *B, float *C, const int M, const int N, const int K);

// Host functions
void MM_DEVICE_GM(const float *A, const float *B, float *C, const int M, const int N, const int K);
void MM_DEVICE_SM(const float *A, const float *B, float *C, const int M, const int N, const int K);
void MM_DEVICE_SM_MWPT(const float *A, const float *B, float *C, const int M, const int N, const int K);
void MM_DEVICE_TC_GM(__half *A, __half *B, float *C, const int M, const int N, const int K);
void MM_DEVICE_TC_SM(__half *A, __half *B, float *C, const int M, const int N, const int K);
void MM_DEVICE_CUBLAS_CC(const float *A, const float *B, float *C, const int M, const int N, const int K);
void MM_DEVICE_CUBLAS_TC(const __half *A, const __half *B, float *C, const int M, const int N, const int K);
void MM_HOST_DOUBLE(const float *A, const float *B, float *C, const int M, const int N, const int K);
double compute_avg_relative_err(const float *A, const float *B, int M, int N);
double find_max_relative_err(const float *A, const float *B, int M, int N);

// Function pointer type for matrix multiplication device functions
typedef void (*MM_DEVICE_FUNC)(const void *, const void *, float *, int, int, int);

// Handle for cuBLAS
hipblasHandle_t handle_global;

int main(void)
{
    float *A, *B, *C;
    __half *A_hf, *B_hf;
    int m, n, k;

    // Read input matrices from binary files
    fread_matrix(FILE_A, &A, &m, &k);
    fread_matrix(FILE_B, &B, &k, &n);
    fread_matrix(FILE_A_HF, &A_hf, &m, &k);
    fread_matrix(FILE_B_HF, &B_hf, &k, &n);
    C = (float *)malloc(m * n * sizeof(float));

    printf("\nM: %d, N: %d, K: %d\n\n", m, n, k);

    // Copy FP32 matrices to device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, m * n * sizeof(float));

    // Copy __half matrices to device memory
    __half *d_A_hf, *d_B_hf;

    hipMalloc(&d_A_hf, m * k * sizeof(__half));
    hipMalloc(&d_B_hf, k * n * sizeof(__half));

    hipMemcpy(d_A_hf, A_hf, m * k * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_B_hf, B_hf, k * n * sizeof(__half), hipMemcpyHostToDevice);

    // Function pointers to the corresponding matrix multiplication implementations
    MM_DEVICE_FUNC funcs[] = {
        (MM_DEVICE_FUNC)MM_DEVICE_GM,
        (MM_DEVICE_FUNC)MM_DEVICE_SM,
        (MM_DEVICE_FUNC)MM_DEVICE_SM_MWPT,
        (MM_DEVICE_FUNC)MM_DEVICE_TC_GM,
        (MM_DEVICE_FUNC)MM_DEVICE_TC_SM,
        (MM_DEVICE_FUNC)MM_DEVICE_CUBLAS_CC,
        (MM_DEVICE_FUNC)MM_DEVICE_CUBLAS_TC,
    };

    // Label each functions
    const char *func_types[] = {
        "CUDA Cores/float",
        "CUDA Cores/float/shared memory",
        "CUDA Cores/float/shared memory/MWPT",
        "Tensor Cores/half",
        "Tensor Cores/half/shared memory",
        "CUDA Cores/float/cuBLAS",
        "Tensor Cores/half/cuBLAS",
    };

    // Output file names for saving result matrices corresponding to each method
    const char *output_files[] = {
        FILE_C_1, FILE_C_2, FILE_C_3, FILE_C_4, FILE_C_5, FILE_C_6, FILE_C_7};

    // Device pointers to input matrices A and B
    const void *d_A_list[] = {d_A, d_A, d_A, d_A_hf, d_A_hf, d_A, d_A_hf};
    const void *d_B_list[] = {d_B, d_B, d_B, d_B_hf, d_B_hf, d_B, d_B_hf};

    hipblasCreate(&handle_global);

    double total_time_ms[7] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
    // Test seven matrix multiplication functions
    for (int i = 0; i < 7; i++)
    {
        // Initialize output matrix C to zero on device memory
        hipMemset(d_C, 0, m * n * sizeof(float));

        // Warm-up
        for (int iter = 0; iter < 10; iter++)
        {
            funcs[i](d_A_list[i], d_B_list[i], d_C, m, n, k);
        }

        // Repeat NUM_REPEATS times to measure average execution time
        for (int iter = 0; iter < NUM_REPEATS; iter++)
        {
            CHECK_TIME_START(_start);
            funcs[i](d_A_list[i], d_B_list[i], d_C, m, n, k);
            CHECK_TIME_END(_start, _end, _compute_time);
            total_time_ms[i] += _compute_time;
        }

        // Copy result matrix from device to host and save it to a file
        hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);
        fwrite_matrix(output_files[i], C, m, n);
    }

    // Run host-side matrix multiplication using double precision for reference
    MM_HOST_DOUBLE(A, B, C, m, n, k);

    for (int i = 0; i < 7; i++)
    {
        float *C_result;
        int m_result, n_result;

        // Compute average execution time over NUM_REPEATS
        double avg_time = total_time_ms[i] / NUM_REPEATS;

        // Read result matrix generated by the i-th GPU method from file
        fread_matrix(output_files[i], &C_result, &m_result, &n_result);

        // Compute average and maximum absolute relative errors between GPU and CPU results
        double relative_err = compute_avg_relative_err(C, C_result, m_result, n_result);
        double max_err = find_max_relative_err(C, C_result, m_result, n_result);

        // Print results
        printf("[%d] GPU time(%s) = %e(ms)\n\n", i + 1, func_types[i], avg_time);
        printf("\t[Absolute relative errors] average = %e, max = %e\n\n", relative_err, max_err);

        free(C_result);
    }

    // Destory cuBLAS handle
    hipblasDestroy(handle_global);

    // Free allocated memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_A_hf);
    hipFree(d_B_hf);

    free(A);
    free(B);
    free(A_hf);
    free(B_hf);

    return 0;
}

// ================================================================================================================
// File read/write functions
// ================================================================================================================
void fread_matrix(const char *filename, float **matrix, int *row, int *col)
{
    FILE *fp = fopen(filename, "rb");
    if (fp == NULL)
    {
        printf("fread_matrix Failed!\n");
        exit(1);
    }
    else
    {
        fread(row, sizeof(int), 1, fp);
        fread(col, sizeof(int), 1, fp);
        *matrix = (float *)malloc(*row * *col * sizeof(float));
        fread(*matrix, sizeof(float), *row * *col, fp);
    }
    fclose(fp);
}

void fread_matrix(const char *filename, __half **matrix, int *row, int *col)
{
    FILE *fp = fopen(filename, "rb");
    if (fp == NULL)
    {
        printf("fread_matrix Failed! %s\n", filename);
        exit(1);
    }
    else
    {
        fread(row, sizeof(int), 1, fp);
        fread(col, sizeof(int), 1, fp);
        *matrix = (__half *)malloc(*row * *col * sizeof(__half));
        fread(*matrix, sizeof(__half), *row * *col, fp);
    }
    fclose(fp);
}

void fwrite_matrix(const char *filename, float *matrix, int row, int col)
{
    FILE *fp = fopen(filename, "wb");
    if (fp == NULL)
    {
        printf("fwrite_matrix Failed!\n");
        exit(1);
    }
    else
    {
        fwrite(&row, sizeof(int), 1, fp);
        fwrite(&col, sizeof(int), 1, fp);
        fwrite(matrix, sizeof(float), row * col, fp);
    }
    fclose(fp);
}

// ================================================================================================================
// Matrix multiplication kernels
// ================================================================================================================
__global__ void mm_naive_cc(const float *A, const float *B, float *C, const int M, const int N, const int K)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int t_m = tid / N;
    int t_n = tid % N;

    if (t_m >= M || t_n >= N)
        return;

    float accum = 0.0f;
    for (int k = 0; k < K; k++)
    {
        accum += A[t_m * K + k] * B[k * N + t_n];
    }
    C[t_m * N + t_n] = accum;
}

template <const uint TS>
__global__ void mm_sm_cc(const float *A, const float *B, float *C, const int M, const int N, const int K)
{
    __shared__ float tile_A[TS * TS];
    __shared__ float tile_B[TS * TS];

    int tile_row = blockIdx.x / CEIL_DIV(N, TS);
    int tile_col = blockIdx.x % CEIL_DIV(N, TS);

    int tid = threadIdx.x;
    int local_row = threadIdx.x / TS;
    int local_col = threadIdx.x % TS;

    int global_row = tile_row * TS + local_row;
    int global_col = tile_col * TS + local_col;

    if (global_row >= M || global_col >= N)
        return;

    float accum = 0.0f;
    int A_row_idx = global_row;
    int B_col_idx = global_col;
    for (int k = 0; k < K; k += TS)
    {
        int A_col_idx = k + local_col;
        int B_row_idx = k + local_row;

        tile_A[tid] = (A_col_idx < K) ? A[A_row_idx * K + A_col_idx] : 0.0f;
        tile_B[local_col * TS + local_row] = (B_row_idx < K) ? B[B_row_idx * N + B_col_idx] : 0.0f;

        __syncthreads();

        for (int i = 0; i < TS; i++)
        {
            accum += tile_A[local_row * TS + i] * tile_B[local_col * TS + i];
        }
        __syncthreads();
    }
    C[global_row * N + global_col] = accum;
}

template <const uint TS, const uint WPT, const uint RTS>
__global__ void mm_sm_mwpt_cc(const float *A, const float *B, float *C, const int M, const int N, const int K)
{
    __shared__ float tile_A[TS * TS];
    __shared__ float tile_B[TS * TS];
    float accum[WPT];
    for (int i = 0; i < WPT; i++)
    {
        accum[i] = 0.0f;
    }

    int tile_row = blockIdx.x / CEIL_DIV(N, TS);
    int tile_col = blockIdx.x % CEIL_DIV(N, TS);

    int tid = threadIdx.x;
    int local_row = threadIdx.x / TS;
    int local_col = threadIdx.x % TS;

    int global_row = tile_row * TS + local_row;
    int global_col = tile_col * TS + local_col;

    if (global_row >= M || global_col >= N)
        return;

    for (int k = 0; k < K; k += TS)
    {
        int A_col_idx = k + local_col;
        int B_row_idx = k + local_col;

        for (int w = 0; w < WPT; w++)
        {
            int A_row_idx = global_row + w * RTS;
            int B_col_idx = tile_col * TS + w * RTS + local_row;

            tile_A[tid + w * RTS * TS] = (A_col_idx < K) ? A[A_row_idx * K + A_col_idx] : 0.0f;
            tile_B[tid + w * RTS * TS] = (B_row_idx < K) ? B[B_row_idx * N + B_col_idx] : 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TS; i++)
        {
            float tmp = tile_B[local_col * TS + i];
            for (int w = 0; w < WPT; w++)
            {
                accum[w] += tile_A[w * RTS * TS + local_row * TS + i] * tmp;
            }
        }
    }
    for (int w = 0; w < WPT; w++)
    {
        C[global_row * N + global_col + w * RTS * N] = accum[w];
    }
}

template <const uint WMMA_M, const uint WMMA_N, const uint WMMA_K>
__global__ void mm_naive_tc(__half *A, __half *B, float *C, const int M, const int N, const int K)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int wid = tid / warpSize;
    int warp_row = wid / CEIL_DIV(N, WMMA_N);
    int warp_col = wid % CEIL_DIV(N, WMMA_N);

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    for (int k = 0; k < K; k += WMMA_K)
    {
        __half *A_start = A + warp_row * WMMA_M * K + k;
        __half *B_start = B + k * N + warp_col * WMMA_N;
        wmma::load_matrix_sync(a_frag, A_start, K);
        wmma::load_matrix_sync(b_frag, B_start, N);

        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
    float *C_start = C + warp_row * WMMA_M * N + warp_col * WMMA_N;
    wmma::store_matrix_sync(C_start, c_frag, N, wmma::mem_row_major);
}

template <const uint TILE_M, const uint TILE_N, const uint TILE_K, const uint WMMA_M, const uint WMMA_N, const uint WMMA_K>
__global__ void mm_sm_tc(__half *A, __half *B, float *C, const int M, const int N, const int K)
{

    __shared__ __align__(128) __half shared_memory[TILE_M * TILE_K + TILE_K * TILE_N];

    __half *A_shared = shared_memory;
    __half *B_shared = shared_memory + TILE_M * TILE_K;

    int tile_row = blockIdx.x / CEIL_DIV(N, TILE_N);
    int tile_col = blockIdx.x % CEIL_DIV(N, TILE_N);

    int wid = threadIdx.x / warpSize;
    int wrow = wid / CEIL_DIV(TILE_N, WMMA_N);
    int wcol = wid % CEIL_DIV(TILE_N, WMMA_N);

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    for (int k = 0; k < K; k += TILE_K)
    {
        for (int i = threadIdx.x; i < TILE_M * TILE_K; i += blockDim.x)
        {
            A_shared[i] = A[(tile_row * TILE_M + (i / TILE_K)) * K + (i % TILE_K) + k];
        }
        for (int i = threadIdx.x; i < TILE_K * TILE_N; i += blockDim.x)
        {
            B_shared[i] = B[(k + (i / TILE_N)) * N + tile_col * TILE_N + (i % TILE_N)];
        }
        __syncthreads();

        for (int i = 0; i < TILE_K; i += WMMA_K)
        {
            wmma::load_matrix_sync(a_frag, &A_shared[wrow * WMMA_M * TILE_K + i], TILE_K);
            wmma::load_matrix_sync(b_frag, &B_shared[i * TILE_N + wcol * WMMA_N], TILE_N);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            __syncthreads();
        }
    }
    wmma::store_matrix_sync(&C[(tile_row * TILE_M + wrow * WMMA_M) * N + tile_col * TILE_N + wcol * WMMA_N], c_frag, N, wmma::mem_row_major);
}

// ================================================================================================================
// Host Functions
// ================================================================================================================
void MM_DEVICE_GM(const float *A, const float *B, float *C, const int M, const int N, const int K)
{

    int size_C = M * N;
    dim3 blockDim(256);
    dim3 gridDim(CEIL_DIV(size_C, blockDim.x));
    mm_naive_cc<<<gridDim, blockDim>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}

void MM_DEVICE_SM(const float *A, const float *B, float *C, const int M, const int N, const int K)
{
    const uint TS = 32;
    dim3 blockDim(TS * TS);
    dim3 gridDim(CEIL_DIV(M, TS) * CEIL_DIV(N, TS));
    mm_sm_cc<TS><<<gridDim, blockDim>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}

void MM_DEVICE_SM_MWPT(const float *A, const float *B, float *C, const int M, const int N, const int K)
{
    const uint TS = 16;
    const uint WPT = 8;
    const uint RTS = TS / WPT;
    dim3 blockDim(TS * RTS);
    dim3 gridDim(CEIL_DIV(M, TS) * CEIL_DIV(N, TS));
    mm_sm_mwpt_cc<TS, WPT, RTS><<<gridDim, blockDim>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}

void MM_DEVICE_TC_GM(__half *A, __half *B, float *C, const int M, const int N, const int K)
{
    const uint WMMA_M = 16;
    const uint WMMA_N = 16;
    const uint WMMA_K = 16;
    dim3 blockDim(256);
    int warp_per_block = blockDim.x / 32;
    dim3 gridDim(CEIL_DIV(CEIL_DIV(M, WMMA_M) * CEIL_DIV(N, WMMA_N), warp_per_block));
    mm_naive_tc<WMMA_M, WMMA_N, WMMA_K><<<gridDim, blockDim>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}

void MM_DEVICE_TC_SM(__half *A, __half *B, float *C, const int M, const int N, const int K)
{
    const uint TILE_M = 64;
    const uint TILE_N = 32;
    const uint TILE_K = 16;
    const uint WMMA_M = 16;
    const uint WMMA_N = 16;
    const uint WMMA_K = 16;
    const uint shm_size = TILE_M * TILE_K + TILE_K * TILE_N;
    dim3 blockDim((TILE_M / WMMA_M * TILE_N / WMMA_N) * 32);
    dim3 gridDim(CEIL_DIV(M, TILE_M) * CEIL_DIV(N, TILE_N));

    mm_sm_tc<TILE_M, TILE_N, TILE_K, WMMA_M, WMMA_N, WMMA_K><<<gridDim, blockDim, shm_size>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}

void MM_DEVICE_CUBLAS_CC(const float *A, const float *B, float *C, const int M, const int N, const int K)
{
    float alpha = 1.0, beta = 0.0;

    hipblasGemmEx(handle_global, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F, N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
}

void MM_DEVICE_CUBLAS_TC(const __half *A, const __half *B, float *C, const int M, const int N, const int K)
{
    float alpha = 1.0, beta = 0.0;
    hipblasGemmEx(handle_global, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_16F, N, A, HIP_R_16F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
}

void MM_HOST_DOUBLE(const float *A, const float *B, float *C, const int M, const int N, const int K)
{
    for (int m = 0; m < M; m++)
    {
        for (int n = 0; n < N; n++)
        {
            double accum = 0.0;
            for (int k = 0; k < K; k++)
            {
                accum += A[m * K + k] * B[k * N + n];
            }
            C[m * N + n] = (float)accum;
        }
    }
}

double compute_avg_relative_err(const float *A, const float *B, int M, int N)
{
    double error = 0.0f;

    for (int i = 0; i < M * N; ++i)
    {
        double diff = fabs(A[i] - B[i] / A[i]);
        error += diff;
    }

    return error / (M * N);
}

double find_max_relative_err(const float *A, const float *B, int M, int N)
{
    double max_err = 0.0f;

    for (int i = 0; i < M * N; ++i)
    {
        double diff = fabs(A[i] - B[i] / A[i]);
        if (diff > max_err)
        {
            max_err = diff;
        }
    }

    return max_err;
}